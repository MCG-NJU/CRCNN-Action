#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the license found in the
 * LICENSE file in the root directory of this source tree.
 */
 /**
  * based on:
  * Copyright (c) 2016-present, Facebook, Inc.
  *
  * Licensed under the Apache License, Version 2.0 (the "License");
  * you may not use this file except in compliance with the License.
  * You may obtain a copy of the License at
  *
  *     http://www.apache.org/licenses/LICENSE-2.0
  *
  * Unless required by applicable law or agreed to in writing, software
  * distributed under the License is distributed on an "AS IS" BASIS,
  * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
  * See the License for the specific language governing permissions and
  * limitations under the License.
  */
  
#include "caffe2/video/affine_nd_op.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {
template <typename T>
__global__ void ScaleBiasForwardNd(
    const int n,
    const T* in,
    const T* scale,
    const T* bias,
    const int scale_dim,
    const int hxw_dim,
    T* out) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int scale_index = (index / hxw_dim) % scale_dim;
    out[index] = in[index] * scale[scale_index] + bias[scale_index];
  }
}

template <typename T>
__global__ void ScaleForwardNd(
    const int n,
    const T* in,
    const T* scale,
    const int scale_dim,
    const int hxw_dim,
    T* out) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int scale_index = (index / hxw_dim) % scale_dim;
    out[index] = in[index] * scale[scale_index];
  }
}
} // namespace

template <>
bool AffineNdOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& scale = Input(1);
  auto& bias = Input(2);
  auto* Y = Output(0);

  Y->ResizeLike(X);
  const int output_size = Y->size();
  ScaleBiasForwardNd<float>
      <<<CAFFE_GET_BLOCKS(output_size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          scale.data<float>(),
          bias.data<float>(),
          X.dim32(1),
          X.size() / X.dim32(0) / X.dim32(1),  // support TxHxW
          Y->mutable_data<float>());
  return true;
}

template <>
bool AffineNdGradientOp<float, HIPContext>::RunOnDevice() {
  auto& scale = Input(0);
  auto& dY = Input(1);
  auto* dX = Output(0);

  dX->ResizeLike(dY);
  ScaleForwardNd<float>
      <<<CAFFE_GET_BLOCKS(dY.size()),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          dY.size(),
          dY.data<float>(),
          scale.data<float>(),
          dY.dim32(1),
          dY.size() / dY.dim32(0) / dY.dim32(1),  // support TxHxW
          dX->mutable_data<float>());
  return true;
}

REGISTER_HIP_OPERATOR(AffineNd, AffineNdOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    AffineNdGradient,
    AffineNdGradientOp<float, HIPContext>);
} // namespace caffe2
